#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <math.h>

#define RANGE 11.79


__global__ void vecGPU(float*, float*, float*, int);



int main(int argc, char *argv[]){

	int n = 0; //number of elements in the arrays
	int i;  //loop index
	float *a, *b, *c; // The arrays that will be processed in the host.
	float *temp;  //array in host used in the sequential code.
	float *ad, *bd, *cd; //The arrays that will be processed in the device.
	clock_t start, end; // to meaure the time taken by a specific part of code
	
	if(argc != 2){
		printf("usage:  ./vectorprog n\n");
		printf("n = number of elements in each vector\n");
		exit(1);
		}
		
	n = atoi(argv[1]);
	printf("Each vector will have %d elements\n", n);
	
	
	//Allocating the arrays in the host
	
	if( !(a = (float *)malloc(n*sizeof(float))) )
	{
	   printf("Error allocating array a\n");
	   exit(1);
	}
	
	if( !(b = (float *)malloc(n*sizeof(float))) )
	{
	   printf("Error allocating array b\n");
	   exit(1);
	}
	
	if( !(c = (float *)malloc(n*sizeof(float))) )
	{
	   printf("Error allocating array c\n");
	   exit(1);
	}
	
	if( !(temp = (float *)malloc(n*sizeof(float))) )
	{
	   printf("Error allocating array temp\n");
	   exit(1);
	}
	
	
	srand((unsigned int)time(NULL));
	for (i = 0; i < n;  i++){
        a[i] = ((float)rand()/(float)(RAND_MAX)) * RANGE;
		b[i] = ((float)rand()/(float)(RAND_MAX)) * RANGE;
		c[i] = ((float)rand()/(float)(RAND_MAX)) * RANGE;
		temp[i] = c[i]; //temp is just another copy of C
	}
	
    //The sequential part
	start = clock();
	for(i = 0; i < n; i++)
		temp[i] += a[i] * b[i];
	end = clock();
	printf("Total time taken by the sequential part = %lf\n", (double)(end - start) / CLOCKS_PER_SEC);


	//1. allocate ad, bd, and cd in the device
	int size = n*sizeof(float);
	hipMalloc((void**)&ad, size);
	hipMalloc((void**)&bd, size);
	hipMalloc((void**)&cd, size);

	//2. sned a, b and c to the device
	hipMemcpy(ad, a, size, hipMemcpyHostToDevice);
	hipMemcpy(bd, b, size, hipMemcpyHostToDevice);
	hipMemcpy(cd, c, size, hipMemcpyHostToDevice);

	//3. write the kernel, call it: vecGPU

	//start time
	start = clock();

	//4. Call the kernel
	vecGPU<<<4, 500>>>(ad, bd, cd, n);

	end = clock();
	
	//5. Bring the cd array back from the device and store it in c array (declared earlier in main)
	hipMemcpy(c, cd, size, hipMemcpyDeviceToHost);

	//6. free ad, bd, and cd
	hipFree(ad);
	hipFree(bd);
	hipFree(cd);
	
	printf("Total time taken by the GPU part = %lf\n", (double)(end - start) / CLOCKS_PER_SEC);
	
	//checking the correctness of the GPU part
	for(i = 0; i < n; i++)
	  if( fabsf(temp[i] - c[i]) >= 0.009) //compare up to the second degit in floating point
		printf("Element %d in the result array does not match the sequential version\n", i);
		
	// Free the arrays in the host
	free(a); free(b); free(c); free(temp);

	return 0;
}


__global__ void vecGPU(float* ad, float* bd, float* cd, int n) {
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	int interval = blockDim.x * gridDim.x;
	int j = i;
	while (j < n) {
		cd[j] += ad[j] * bd[j];
		j += interval;
	}
}